#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#include <stdio.h>
#include <math.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>



////////////////////////////////////////////////////////////////////////////////


#include "jpeglib.h"

/*
 * IMAGE DATA FORMATS:
 *
 * The standard input image format is a rectangular array of pixels, with
 * each pixel having the same number of "component" values (color channels).
 * Each pixel row is an array of JSAMPLEs (which typically are unsigned chars).
 * If you are working with color data, then the color values for each pixel
 * must be adjacent in the row; for example, R,G,B,R,G,B,R,G,B,... for 24-bit
 * RGB color.
 */

/* The "frame structure" structure contains an image frame (in RGB or grayscale
 * formats) for passing around the CS338 projects.
 */
typedef struct frame_struct
{
  JSAMPLE *image_buffer;	/* Points to large array of R,G,B-order/grayscale data
                             * Access directly with:
                             *   image_buffer[num_components*pixel + component]
                             */
  JSAMPLE **row_pointers;	/* Points to an array of pointers to the beginning
                             * of each row in the image buffer.  Use to access
                             * the image buffer in a row-wise fashion, with:
                             *   row_pointers[row][num_components*pixel + component]
                             */
  int image_height;		/* Number of rows in image */
  int image_width;		/* Number of columns in image */
  int num_components;	/* Number of components (usually RGB=3 or gray=1) */
} frame_struct_t;
typedef frame_struct_t *frame_ptr;




#define MAXINPUTS 1
#define MAXOUTPUTS 1
frame_ptr input_frames[MAXINPUTS];	/* Pointers to input frames */
frame_ptr output_frames[MAXOUTPUTS];	/* Pointers to output frames */

/* Read/write JPEGs, for program startup & shutdown */
/* YOU SHOULD NOT NEED TO USE THESE AT ALL */
void write_JPEG_file (char * filename, frame_ptr p_info, int quality);
frame_ptr read_JPEG_file (char * filename);

/* Allocate/deallocate frame buffers, USE AS NECESSARY! */
frame_ptr allocate_frame(int height, int width, int num_components);
void destroy_frame(frame_ptr kill_me);

/*
 * write_JPEG_file writes out the contents of an image buffer to a JPEG.
 * A quality level of 2-100 can be provided (default = 75, high quality = ~95,
 * low quality = ~25, utter pixellation = 2).  Note that unlike read_JPEG_file,
 * it does not do any memory allocation on the buffer passed to it.
 */

void write_JPEG_file (char * filename, frame_ptr p_info, int quality)
{
  struct jpeg_compress_struct cinfo;
  struct jpeg_error_mgr jerr;
  FILE * outfile;		/* target file */

  /* Step 1: allocate and initialize JPEG compression object */
  cinfo.err = jpeg_std_error(&jerr);
  jpeg_create_compress(&cinfo);

  /* Step 2: specify data destination (eg, a file) */
  /* Note: steps 2 and 3 can be done in either order. */

  if ((outfile = fopen(filename, "wb")) == NULL) {
    fprintf(stderr, "ERROR: Can't open output file %s\n", filename);
    exit(1);
  }
  jpeg_stdio_dest(&cinfo, outfile);

  /* Step 3: set parameters for compression */

  /* Set basic picture parameters (not optional) */
  cinfo.image_width = p_info->image_width; 	/* image width and height, in pixels */
  cinfo.image_height = p_info->image_height;
  cinfo.input_components = p_info->num_components; /* # of color components per pixel */
  if (p_info->num_components == 3)
    cinfo.in_color_space = JCS_RGB; 	/* colorspace of input image */
  else if (p_info->num_components == 1)
    cinfo.in_color_space = JCS_GRAYSCALE;
  else {
    fprintf(stderr, "ERROR: Non-standard colorspace for compressing!\n");
    exit(1);
  }
  /* Fill in the defaults for everything else, then override quality */
  jpeg_set_defaults(&cinfo);
  jpeg_set_quality(&cinfo, quality, TRUE /* limit to baseline-JPEG values */);

  /* Step 4: Start compressor */
  jpeg_start_compress(&cinfo, TRUE);

  /* Step 5: while (scan lines remain to be written) */
  /*           jpeg_write_scanlines(...); */
  while (cinfo.next_scanline < cinfo.image_height) {
    (void) jpeg_write_scanlines(&cinfo, &(p_info->row_pointers[cinfo.next_scanline]), 1);
  }

  /* Step 6: Finish compression & close output */

  jpeg_finish_compress(&cinfo);
  fclose(outfile);

  /* Step 7: release JPEG compression object */
  jpeg_destroy_compress(&cinfo);
}


/*
 * read_JPEG_file reads the contents of a JPEG into an image buffer, which
 * is automatically allocated after the size of the image is determined.
 * We want to return a frame struct on success, NULL on error.
 */

frame_ptr read_JPEG_file (char * filename)
{
  /* This struct contains the JPEG decompression parameters and pointers to
   * working space (which is allocated as needed by the JPEG library).
   */
  struct jpeg_decompress_struct cinfo;
  struct jpeg_error_mgr jerr;
  FILE * infile;		/* source file */
  frame_ptr p_info;		/* Output frame information */

  //  JSAMPLE *realBuffer;
  //  JSAMPLE **buffer;		/* Output row buffer */
  //  int row_stride;		/* physical row width in output buffer */

  /* Step 1: allocate and initialize JPEG decompression object */
  cinfo.err = jpeg_std_error(&jerr);
  jpeg_create_decompress(&cinfo);

  /* Step 2: open & specify data source (eg, a file) */
  if ((infile = fopen(filename, "rb")) == NULL) {
    fprintf(stderr, "ERROR: Can't open input file %s\n", filename);
    exit(1);
  }
  jpeg_stdio_src(&cinfo, infile);

  /* Step 3: read file parameters with jpeg_read_header() */
  (void) jpeg_read_header(&cinfo, TRUE);

  /* Step 4: use default parameters for decompression */

  /* Step 5: Start decompressor */
  (void) jpeg_start_decompress(&cinfo);

  /* Step X: Create a frame struct & buffers and fill in the blanks */
  fprintf(stderr, "  Opened %s: height = %d, width = %d, c = %d\n",
      filename, cinfo.output_height, cinfo.output_width, cinfo.output_components);
  p_info = allocate_frame(cinfo.output_height, cinfo.output_width, cinfo.output_components);

  /* Step 6: while (scan lines remain to be read) */
  /*           jpeg_read_scanlines(...); */
  while (cinfo.output_scanline < cinfo.output_height) {
    (void) jpeg_read_scanlines(&cinfo, &(p_info->row_pointers[cinfo.output_scanline]), 1);
  }

  /* Step 7: Finish decompression */
  (void) jpeg_finish_decompress(&cinfo);

  /* Step 8: Release JPEG decompression object & file */
  jpeg_destroy_decompress(&cinfo);
  fclose(infile);

  /* At this point you may want to check to see whether any corrupt-data
   * warnings occurred (test whether jerr.pub.num_warnings is nonzero).
   */

  /* And we're done! */
  return p_info;
}


/*
 * allocate/destroy_frame allocate a frame_struct_t and fill in the
 *  blanks appropriately (including allocating the actual frames), and
 *  then destroy them afterwards.
 */

frame_ptr allocate_frame(int height, int width, int num_components)
{
  int row_stride;		/* physical row width in output buffer */
  int i;
  frame_ptr p_info;		/* Output frame information */

  /* JSAMPLEs per row in output buffer */
  row_stride = width * num_components;

  /* Basic struct and information */
  if ((p_info = (frame_struct_t*)malloc(sizeof(frame_struct_t))) == NULL) {
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }
  p_info->image_height = height;
  p_info->image_width = width;
  p_info->num_components = num_components;

  /* Image array and pointers to rows */
  if ((p_info->row_pointers = (JSAMPLE**)malloc(sizeof(JSAMPLE *) * height)) == NULL) {
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }
  if ((p_info->image_buffer = (JSAMPLE*)malloc(sizeof(JSAMPLE) * row_stride * height)) == NULL) {
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }
  for (i=0; i < height; i++)
  	p_info->row_pointers[i] = & (p_info->image_buffer[i * row_stride]);

  /* And send it back! */
  return p_info;
}

void destroy_frame(frame_ptr kill_me)
{
	free(kill_me->image_buffer);
	free(kill_me->row_pointers);
	free(kill_me);
}


void usage()
{
  fprintf(stderr, "ERROR: Need to specify input file and then output file\n");
  exit(1);
}


/* Makes sure values match in the two images*/
void checkResults(frame_ptr f1, frame_ptr f2)
{
  int i, j, k;

  if(f1->image_height != f2->image_height && f1->image_width != f2->image_width
		&& f1->num_components != f2->num_components){
	fprintf(stderr, "Dimensions do not match\n");
	exit(1);
  }

  for (i=0; i < f1->image_height; i++){
    for (j=0; j < f1->image_width; j++){
      for (k=0; k < f1->num_components; k++){
		JSAMPLE j1 = f1->row_pointers[i][(f1->num_components)*j+k];
		JSAMPLE j2 = f2->row_pointers[i][(f2->num_components)*j+k];
		if(j1 != j2){
			fprintf(stderr, "Values do not match at (%d, %d, %d) \n", i, j, k);
			fprintf(stderr, "from %d\n", j1);
			fprintf(stderr, "to %d\n", j2);
			exit(1);
		}
      }
    }
  }

}

void runKernel(frame_ptr input, frame_ptr result);

void uni_blur(frame_ptr from, frame_ptr to) {
  int width = from->image_width;
  int height = from->image_height;

  int max_dimension = height > width ?
                        height : width;
  float percent_blur = 0.05;
  int blur_radius = percent_blur * max_dimension;

  for (int row = 0; row < height; row++)
    for (int column = 0; column < width; column++)
      for (int k = 0; k < from->num_components; k++) {
        int numerator = 0;
        int denominator = 0;
        for (int i = -(blur_radius - 1); i < blur_radius; i++) {
          // if out of bounds
          if (i + column < 0 || i + column >= width)
            continue;

            for (int j = -(blur_radius - 1); j < blur_radius; j++) {
              // if out of bounds
              if(j + row < 0 || j + row >= height)
                   continue;
              numerator += from->row_pointers[row + j][(from->num_components) * (column + i) + k] *
                           (blur_radius - abs(i)) * (blur_radius - abs(j));

              denominator += (blur_radius - abs(i)) * (blur_radius - abs(j));
            }

        }
        int average = numerator/denominator;
        to->row_pointers[row][(from->num_components)*column+k] = average;
      }
}


/*
 * This is just a helper method. It should call runKernel to set up and
 * invoke the kernel.  It should then also call the uniprocessor version
 * of your blurring code (which does not need to be optimized) and
 * check for correctness of your kernel code.
 */
void
runTest( int argc, char** argv)
{

  frame_ptr from = input_frames[0];
  // Allocate frame for kernel to store its results into
  output_frames[0] = allocate_frame(from->image_height, from->image_width, from->num_components);

  frame_ptr to = output_frames[0];

  // initialize output array to all black
  for (int i=0; i < from->image_height; i++)
    for (int j = 0; j < from->image_width * from->num_components; j++)
      to->row_pointers[i][j] = 0;

  // call kernel
  runKernel(from, to);

  // invoke uniprocessor version and check results of kernel to uniprocessor
  // version
  // frame_ptr uni_to = allocate_frame(from->image_height, from->image_width, from->num_components);
  // uni_blur(from, uni_to);
  // checkResults(to, uni_to);

}

/* Turns i and j displacement into single dimension index for the flat array
 * of weights.
 *
 * i and j must be within [-(r-1), r-1]
 *
 * params
 * r blur radius
 * i,j i displacement
 */
__device__ int index_to_weights(int r, int i, int j) {
  int row = i - 1 + r;
  int column = j - 1 + r;
  return row * ((2 * (r-1)) + 1) + column;
}

/* Returns an array of precomputed the weights
 *
 * size must represent the cardinality of [-(r - 1), -1] U [0] U [1, r-1]
 *
 * params
 * r Blur radius
 * size Size of weight array
 */
int* precompute_weights(int r, int size) {
  int* weights = (int*) malloc( size * sizeof(int) );

  int counter = 0;
  for (int i = -(r-1); i < r; i++)
    for (int j = -(r-1); j < r; j++)
      weights[counter++] =  (r - abs(i)) * (r - abs(j));

  return weights;

}

/* Returns division factor
 *
 * params
 * r Blur radius
 */
int compute_division_factor(int r) {
  int division_factor = 0;
  for (int i = -(r-1); i < r; i++)
    for (int j = -(r-1); j < r; j++)
      division_factor += (r - abs(i)) * (r - abs(j));

  return division_factor;
}

/**
 * CUDA Kernel Device code
 * This is code for blurring a single pixel
 *
*/

// naive kernel without branching
__global__ void cs338Blur(unsigned char* from, unsigned char* to,
                          int blur_radius, int height, int width,
                          int num_components, int n)
{
  // computes which block this thread lies on
  int blockId = blockIdx.x + (gridDim.x * blockIdx.y);

  // computes the index of the thread
  int index = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.y) + threadIdx.x;

  if (index < n) {

    // computes values related to 2D frame
    int thread_x = (index / num_components) % width;
    int thread_y = (index / num_components) / width;
    int thread_k = index % num_components;

    int numerator = 0;
    int denominator = 0;
    for (int i = -(blur_radius - 1); i < blur_radius; i++) {
      // if out of bounds
      if(i + thread_y < 0 || i + thread_y >= height)
           continue;

        for (int j = -(blur_radius - 1); j < blur_radius; j++) {
          // if out of bounds
          if (j + thread_x < 0 || j + thread_x >= width)
            continue;

          numerator += from[((thread_y + i) * width * num_components + (thread_x + j) * num_components) + thread_k] *
                       (blur_radius - abs(i)) * (blur_radius - abs(j));

          denominator += (blur_radius - abs(i)) * (blur_radius - abs(j));
        }

    }

    to[index] = numerator/denominator;
  }

}

// kernel that reduces branching
__global__ void cs338Blur2(unsigned char* from, unsigned char* to,
                          int blur_radius, int height, int width,
                          int num_components, int n)
{
  // computes which block this thread lies on
  int blockId = blockIdx.x + (gridDim.x * blockIdx.y);

  // computes the index of the thread
  int index = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.y) + threadIdx.x;

  if (index < n) {

    // computes values related to 2D frame
    int thread_x = (index / num_components) % width;
    int thread_y = (index / num_components) / width;
    int thread_k = index % num_components;

    /* (Thinking in 2D)
     * Within r pixels of the edges of the image we need to go as far back as
     * possible and go as far forward as possible.
     * For starting, we can't go back -(blur_radius - 1) pixels, the most we can
     * do is the edge of the frame: go back -thread_y or -thread_x pixels.
     * For ending, we can't go blur_radius pixels ahead, so we need to go to the
     * edge of the frame: go forward the remaining pixels.
     */
    int row_start = thread_y >= blur_radius ? -(blur_radius - 1) : -thread_y;
    int column_start = thread_x >= blur_radius ? -(blur_radius - 1) : -thread_x;
    int row_finish = thread_y < height - blur_radius ? blur_radius : height - thread_y;
    int column_finish = thread_x < width - blur_radius ? blur_radius : width - thread_x;

    int numerator = 0;
    int denominator = 0;
    for (int i = row_start; i < row_finish; i++) {
        for (int j = column_start; j < column_finish; j++) {

          numerator += from[((thread_y + i) * width * num_components + (thread_x + j) * num_components) + thread_k] *
                       (blur_radius - abs(i)) * (blur_radius - abs(j));

          denominator += (blur_radius - abs(i)) * (blur_radius - abs(j));
        }

    }

    to[index] = numerator/denominator;
  }

}

// kernel that has branches for valid and invalid pixels and where weights and
// division factor are precomputed
__global__ void cs338Blur3(unsigned char* from, unsigned char* to,
                             int blur_radius, int height, int width,
                             int num_components, int n, int* weights,
                             int division_factor)
{
  // computes which block this thread lies on
  int blockId = blockIdx.x + (gridDim.x * blockIdx.y);

  // computes the index of the thread
  int index = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.y) + threadIdx.x;

  if (index < n) {

    // computes values related to 2D frame
    int thread_x = (index / num_components) % width;
    int thread_y = (index / num_components) / width;
    int thread_k = index % num_components;

    // if valid pixels
    if (thread_x >= blur_radius - 1 && thread_x < width - (blur_radius - 1) &&
        thread_y >= blur_radius - 1 && thread_y < height - (blur_radius - 1)) {
          // then use weights array and division factor
          int numerator = 0;
          for (int i = -(blur_radius - 1); i < blur_radius; i++) {
              for (int j = -(blur_radius - 1); j < blur_radius; j++) {

                numerator += from[((thread_y + i) * width * num_components + (thread_x + j) * num_components) + thread_k] *
                             weights[index_to_weights(blur_radius, i, j)];

              }

          }

          to[index] = numerator/division_factor;
    } // else the pixels are around the edge
    else {

      /* (Thinking in 2D)
       * Within r pixels of the edges of the image we need to go as far back as
       * possible and go as far forward as possible.
       * For starting, we can't go back -(blur_radius - 1) pixels, the most we can
       * do is the edge of the frame: go back -thread_y or -thread_x pixels.
       * For ending, we can't go blur_radius pixels ahead, so we need to go to the
       * edge of the frame: go forward the remaining pixels.
       */
      int row_start = thread_y >= blur_radius ? -(blur_radius - 1) : -thread_y;
      int column_start = thread_x >= blur_radius ? -(blur_radius - 1) : -thread_x;
      int row_finish = thread_y < height - blur_radius ? blur_radius : height - thread_y;
      int column_finish = thread_x < width - blur_radius ? blur_radius : width - thread_x;

      int numerator = 0;
      int denominator = 0;
      // still uses weights array
      for (int i = row_start; i < row_finish; i++) {
          for (int j = column_start; j < column_finish; j++) {

            numerator += from[((thread_y + i) * width * num_components + (thread_x + j) * num_components) + thread_k] *
                         weights[index_to_weights(blur_radius, i, j)];

            denominator += (blur_radius - abs(i)) * (blur_radius - abs(j));

          }

      }

      /* Cannot use division factor because I cannot pass in different factors
       * to account for invalid pixels.
       */
      to[index] = numerator/denominator;
    }
  }

}




/**
 * Host main routine
 */
int
main(int argc, char **argv)
{

  if(argc < 3){
    usage();
    exit(1);
  }

  // Load input file
  input_frames[0] = read_JPEG_file(argv[1]);

  // Do the actual work including calling CUDA kernel
  runTest(argc, argv);

  // Write output file
  write_JPEG_file(argv[2], output_frames[0], 75);

  return 0;
}

//********************************************************************************************************************************************


// This sets up GPU device by allocating the required memory and then
// calls the kernel on GPU. (You might choose to add/remove arguments.)
// It's currently set up to use the global variables and write its
// final results into the specified argument.
void
runKernel(frame_ptr input, frame_ptr result)
{

  // calculates blur radius
  int max_dimension = input-> image_height > input->image_width ?
                        input-> image_height : input->image_width;
  float percent_blur = 0.05;
  int blur_radius = percent_blur * max_dimension;

  // size of flattened array
  int size = input->image_height * input->image_width * input->num_components * sizeof(unsigned char);

  unsigned char* input_d; // ptr to input image
  unsigned char* result_d; // ptr to output image

  // 1. Transfer input to device memory
  checkCudaErrors(hipMalloc((void **) &input_d, size));
  checkCudaErrors(hipMemcpy(input_d, input->image_buffer, size, hipMemcpyHostToDevice));

  // Allocate device memory for result
  checkCudaErrors(hipMalloc((void **) &result_d, size));

  // 2. Kernel invocation code
  // 32 is the maximum number for this dimension (32, 16, 8, 4, 2)
  int blockDim = 32;
  dim3 DimGrid(sqrt(size/(blockDim * blockDim)), sqrt(size/(blockDim * blockDim)), 1);
  // takes ceiling of x and y dimensions
  if (sqrt(size/(blockDim * blockDim))) {
    DimGrid.x++;
    DimGrid.y++;
  }
  dim3 DimBlock(blockDim, blockDim, 1);

  // for part 3 we need to precompute the weights and the division factor
  #define PART3
  #ifdef PART3
    int* weights;
    int* weights_d; // ptr to weights in device

    // The 2D array is square and has a row of size of [-(r - 1), -1] U [0] U [1, r-1]
    int weight_size = ((2 * (blur_radius-1)) + 1) * ((2 * (blur_radius-1)) + 1) * sizeof(int);
    weights = precompute_weights(blur_radius, weight_size);
    int division_factor = compute_division_factor(blur_radius);

    checkCudaErrors(hipMalloc((void **) &weights_d, weight_size));
    checkCudaErrors(hipMemcpy(weights_d, weights, weight_size, hipMemcpyHostToDevice));

  #endif

  // Uses hipEvent_t to get timing information
  hipEvent_t start, stop;
  float time;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // cs338Blur<<<DimGrid, DimBlock>>>(input_d, result_d, blur_radius,
  //                                  input-> image_height, input-> image_width,
  //                                  input->num_components, size);
  // cs338Blur2<<<DimGrid, DimBlock>>>(input_d, result_d, blur_radius,
  //                                  input-> image_height, input-> image_width,
  //                                  input->num_components, size);
  #ifdef PART3
  cs338Blur3<<<DimGrid, DimBlock>>>(input_d, result_d, blur_radius,
                                      input-> image_height, input-> image_width,
                                      input->num_components, size, weights_d,
                                      division_factor);
  #endif

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  printf("Time to blur image:  %3.1f ms \n", time);

  // 3. Transfer result from device to host
  checkCudaErrors(hipMemcpy(result->image_buffer, result_d, size, hipMemcpyDeviceToHost));

  // Free device memory for input, result
  hipFree(input_d);
  hipFree(result_d);
  #ifdef PART3
    hipFree(weights_d);
  #endif
}
