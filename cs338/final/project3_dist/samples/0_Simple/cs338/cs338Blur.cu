#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <time.h>


////////////////////////////////////////////////////////////////////////////////


#include "jpeglib.h"

/*
 * IMAGE DATA FORMATS:
 *
 * The standard input image format is a rectangular array of pixels, with
 * each pixel having the same number of "component" values (color channels).
 * Each pixel row is an array of JSAMPLEs (which typically are unsigned chars).
 * If you are working with color data, then the color values for each pixel
 * must be adjacent in the row; for example, R,G,B,R,G,B,R,G,B,... for 24-bit
 * RGB color.
 */

/* The "frame structure" structure contains an image frame (in RGB or grayscale
 * formats) for passing around the CS338 projects.
 */
typedef struct frame_struct
{
  JSAMPLE *image_buffer;	/* Points to large array of R,G,B-order/grayscale data
                             * Access directly with:
                             *   image_buffer[num_components*pixel + component]
                             */
  JSAMPLE **row_pointers;	/* Points to an array of pointers to the beginning
                             * of each row in the image buffer.  Use to access
                             * the image buffer in a row-wise fashion, with:
                             *   row_pointers[row][num_components*pixel + component]
                             */
  int image_height;		/* Number of rows in image */
  int image_width;		/* Number of columns in image */
  int num_components;	/* Number of components (usually RGB=3 or gray=1) */
} frame_struct_t;
typedef frame_struct_t *frame_ptr;




#define MAXINPUTS 1
#define MAXOUTPUTS 1
frame_ptr input_frames[MAXINPUTS];	/* Pointers to input frames */
frame_ptr output_frames[MAXOUTPUTS];	/* Pointers to output frames */

/* Read/write JPEGs, for program startup & shutdown */
/* YOU SHOULD NOT NEED TO USE THESE AT ALL */
void write_JPEG_file (char * filename, frame_ptr p_info, int quality);
frame_ptr read_JPEG_file (char * filename);

/* Allocate/deallocate frame buffers, USE AS NECESSARY! */
frame_ptr allocate_frame(int height, int width, int num_components);
void destroy_frame(frame_ptr kill_me);

/*
 * write_JPEG_file writes out the contents of an image buffer to a JPEG.
 * A quality level of 2-100 can be provided (default = 75, high quality = ~95,
 * low quality = ~25, utter pixellation = 2).  Note that unlike read_JPEG_file,
 * it does not do any memory allocation on the buffer passed to it.
 */

void write_JPEG_file (char * filename, frame_ptr p_info, int quality)
{
  struct jpeg_compress_struct cinfo;
  struct jpeg_error_mgr jerr;
  FILE * outfile;		/* target file */

  /* Step 1: allocate and initialize JPEG compression object */
  cinfo.err = jpeg_std_error(&jerr);
  jpeg_create_compress(&cinfo);

  /* Step 2: specify data destination (eg, a file) */
  /* Note: steps 2 and 3 can be done in either order. */

  if ((outfile = fopen(filename, "wb")) == NULL) {
    fprintf(stderr, "ERROR: Can't open output file %s\n", filename);
    exit(1);
  }
  jpeg_stdio_dest(&cinfo, outfile);

  /* Step 3: set parameters for compression */

  /* Set basic picture parameters (not optional) */
  cinfo.image_width = p_info->image_width; 	/* image width and height, in pixels */
  cinfo.image_height = p_info->image_height;
  cinfo.input_components = p_info->num_components; /* # of color components per pixel */
  if (p_info->num_components == 3)
    cinfo.in_color_space = JCS_RGB; 	/* colorspace of input image */
  else if (p_info->num_components == 1)
    cinfo.in_color_space = JCS_GRAYSCALE;
  else {
    fprintf(stderr, "ERROR: Non-standard colorspace for compressing!\n");
    exit(1);
  }
  /* Fill in the defaults for everything else, then override quality */
  jpeg_set_defaults(&cinfo);
  jpeg_set_quality(&cinfo, quality, TRUE /* limit to baseline-JPEG values */);

  /* Step 4: Start compressor */
  jpeg_start_compress(&cinfo, TRUE);

  /* Step 5: while (scan lines remain to be written) */
  /*           jpeg_write_scanlines(...); */
  while (cinfo.next_scanline < cinfo.image_height) {
    (void) jpeg_write_scanlines(&cinfo, &(p_info->row_pointers[cinfo.next_scanline]), 1);
  }

  /* Step 6: Finish compression & close output */

  jpeg_finish_compress(&cinfo);
  fclose(outfile);

  /* Step 7: release JPEG compression object */
  jpeg_destroy_compress(&cinfo);
}


/*
 * read_JPEG_file reads the contents of a JPEG into an image buffer, which
 * is automatically allocated after the size of the image is determined.
 * We want to return a frame struct on success, NULL on error.
 */

frame_ptr read_JPEG_file (char * filename)
{
  /* This struct contains the JPEG decompression parameters and pointers to
   * working space (which is allocated as needed by the JPEG library).
   */
  struct jpeg_decompress_struct cinfo;
  struct jpeg_error_mgr jerr;
  FILE * infile;		/* source file */
  frame_ptr p_info;		/* Output frame information */

  //  JSAMPLE *realBuffer;
  //  JSAMPLE **buffer;		/* Output row buffer */
  //  int row_stride;		/* physical row width in output buffer */

  /* Step 1: allocate and initialize JPEG decompression object */
  cinfo.err = jpeg_std_error(&jerr);
  jpeg_create_decompress(&cinfo);

  /* Step 2: open & specify data source (eg, a file) */
  if ((infile = fopen(filename, "rb")) == NULL) {
    fprintf(stderr, "ERROR: Can't open input file %s\n", filename);
    exit(1);
  }
  jpeg_stdio_src(&cinfo, infile);

  /* Step 3: read file parameters with jpeg_read_header() */
  (void) jpeg_read_header(&cinfo, TRUE);

  /* Step 4: use default parameters for decompression */

  /* Step 5: Start decompressor */
  (void) jpeg_start_decompress(&cinfo);

  /* Step X: Create a frame struct & buffers and fill in the blanks */
  fprintf(stderr, "  Opened %s: height = %d, width = %d, c = %d\n",
      filename, cinfo.output_height, cinfo.output_width, cinfo.output_components);
  p_info = allocate_frame(cinfo.output_height, cinfo.output_width, cinfo.output_components);

  /* Step 6: while (scan lines remain to be read) */
  /*           jpeg_read_scanlines(...); */
  while (cinfo.output_scanline < cinfo.output_height) {
    (void) jpeg_read_scanlines(&cinfo, &(p_info->row_pointers[cinfo.output_scanline]), 1);
  }

  /* Step 7: Finish decompression */
  (void) jpeg_finish_decompress(&cinfo);

  /* Step 8: Release JPEG decompression object & file */
  jpeg_destroy_decompress(&cinfo);
  fclose(infile);

  /* At this point you may want to check to see whether any corrupt-data
   * warnings occurred (test whether jerr.pub.num_warnings is nonzero).
   */

  /* And we're done! */
  return p_info;
}


/*
 * allocate/destroy_frame allocate a frame_struct_t and fill in the
 *  blanks appropriately (including allocating the actual frames), and
 *  then destroy them afterwards.
 */

frame_ptr allocate_frame(int height, int width, int num_components)
{
  int row_stride;		/* physical row width in output buffer */
  int i;
  frame_ptr p_info;		/* Output frame information */

  /* JSAMPLEs per row in output buffer */
  row_stride = width * num_components;

  /* Basic struct and information */
  if ((p_info = (frame_struct_t*)malloc(sizeof(frame_struct_t))) == NULL) {
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }
  p_info->image_height = height;
  p_info->image_width = width;
  p_info->num_components = num_components;

  /* Image array and pointers to rows */
  if ((p_info->row_pointers = (JSAMPLE**)malloc(sizeof(JSAMPLE *) * height)) == NULL) {
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }
  if ((p_info->image_buffer = (JSAMPLE*)malloc(sizeof(JSAMPLE) * row_stride * height)) == NULL) {
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }
  for (i=0; i < height; i++)
  	p_info->row_pointers[i] = & (p_info->image_buffer[i * row_stride]);

  /* And send it back! */
  return p_info;
}

void destroy_frame(frame_ptr kill_me)
{
	free(kill_me->image_buffer);
	free(kill_me->row_pointers);
	free(kill_me);
}


void usage()
{
  fprintf(stderr, "ERROR: Need to specify input file and then output file\n");
  exit(1);
}


/* Makes sure values match in the two images*/
void checkResults(frame_ptr f1, frame_ptr f2)
{
  int i, j, k;

  if(f1->image_height != f2->image_height && f1->image_width != f2->image_width
		&& f1->num_components != f2->num_components){
	fprintf(stderr, "Dimensions do not match\n");
	exit(1);
  }

  for (i=0; i < f1->image_height; i++){
    for (j=0; j < f1->image_width; j++){
      for (k=0; k < f1->num_components; k++){
		JSAMPLE j1 = f1->row_pointers[i][(f1->num_components)*j+k];
		JSAMPLE j2 = f2->row_pointers[i][(f2->num_components)*j+k];
		if(j1 != j2){
			fprintf(stderr, "Values do not match at (%d, %d, %d) \n", i, j, k);
			fprintf(stderr, "from %d\n", j1);
			fprintf(stderr, "to %d\n", j2);
			exit(1);
		}
      }
    }
  }

}

void runKernel(frame_ptr result);

// input->num_components should be 1 as the input got changed to grayscale
frame_ptr pad_frame(frame_ptr input) {
  int new_height = input->image_height+2;
  int new_width = input->image_width+2;
  frame_ptr padded = allocate_frame(new_height, new_width, input->num_components);

  // write old values into our new padded image
  for (int row = 0; row < input->image_height; row++)
    for (int col = 0; col < input->image_width; col++)
      padded->row_pointers[row+1][col+1] = input->row_pointers[row][col];

  // extends pixels to first and last columns
  for (int row = 1; row < new_height-1; row++) {
    padded->row_pointers[row][0] = padded->row_pointers[row][1];
    padded->row_pointers[row][new_width-1] = padded->row_pointers[row][new_width-2];
  }

  // extends top and bottom rows
  for (int column = 0; column < new_width; column++) {
    padded->row_pointers[0][column] = padded->row_pointers[1][column];
    padded->row_pointers[new_height-1][column] = padded->row_pointers[new_height-2][column];
  }

  return padded;
}

/* Kernel that runs the edge detection algorithm on each output pixel with
 * hard coded sobel filters
 *
 * input has width and height that's two bigger than to's because of the way
 * convolution works
 */
__global__ void register_memory_edge_detection(unsigned char* input, unsigned char* to,
                               int to_size, int to_width, int to_height,
                               int from_width, int from_height) {

  // computes which block this thread lies on
  int blockId = blockIdx.x + (gridDim.x * blockIdx.y);

  // computes the index of the thread
  int index = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.y) + threadIdx.x;

  if (index >= to_size) return;

  // computes values related to param to's 2D frame
  int thread_x = index % to_width;
  int thread_y = index / to_width;

  // frame in original picture needed for convolution
  int frame[3][3] = {
    {input[(thread_y) * from_width + (thread_x)], input[(thread_y) * from_width + (thread_x + 1)], input[(thread_y) * from_width + (thread_x + 2)]},
    {input[(thread_y + 1) * from_width + (thread_x)], input[(thread_y + 1) * from_width + (thread_x + 1)], input[(thread_y + 1) * from_width + (thread_x + 2)]},
    {input[(thread_y + 2) * from_width + (thread_x)], input[(thread_y + 2) * from_width + (thread_x + 1)], input[(thread_y + 2) * from_width + (thread_x + 2)]}
  };

  // kernels are hard coded
  /* x kernel:
   * [-1, 0 , 1]
   * [-2, 0, 2]
   * [-1, 0, 1]
   */
  int x_convolution = frame[0][2] - frame[0][0]
                      + 2 * frame[1][2] - 2 * frame[1][0]
                      + frame[2][2] - frame[2][0];

  /* y kernel:
   * [-1, -2 , -1]
   * [0, 0, 0]
   * [1, 2, 1]
   */
  int y_convolution = - frame[0][0] - 2 * frame[0][1] - frame[0][2]
                      + frame[2][0] + 2 * frame[2][1] + frame[2][2];

  int gradient_magnitude = sqrt((float)((x_convolution * x_convolution) + (y_convolution * y_convolution)));

  to[index] = gradient_magnitude;
}

/* Kernel that runs the edge detection algorithm on each output pixel with
 * sobel filter as input (on global memory)
 *
 * input has width and height that's two bigger than to's because of the way
 * convolution works
 */
__global__ void global_memory_edge_detection(unsigned char* input, unsigned char* to,
                               int to_size, int to_width, int to_height,
                               int from_width, int from_height,
                               int* x_filter, int* y_filter) {

  // computes which block this thread lies on
  int blockId = blockIdx.x + (gridDim.x * blockIdx.y);

  // computes the index of the thread
  int index = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.y) + threadIdx.x;

  if (index >= to_size) return;

  // computes values related to param to's 2D frame
  int thread_x = index % to_width;
  int thread_y = index / to_width;

  // frame in original picture needed for convolution
  // size is 3 by 3 because each sobel filter is 3 by 3
  int frame[3][3] = {
    {input[(thread_y) * from_width + (thread_x)], input[(thread_y) * from_width + (thread_x + 1)], input[(thread_y) * from_width + (thread_x + 2)]},
    {input[(thread_y + 1) * from_width + (thread_x)], input[(thread_y + 1) * from_width + (thread_x + 1)], input[(thread_y + 1) * from_width + (thread_x + 2)]},
    {input[(thread_y + 2) * from_width + (thread_x)], input[(thread_y + 2) * from_width + (thread_x + 1)], input[(thread_y + 2) * from_width + (thread_x + 2)]}
  };

  int x_convolution = 0;
  int filter_index = 0;
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      x_convolution += frame[i][j] * x_filter[filter_index];
      filter_index++;
    }
  }

  int y_convolution = 0;
  filter_index = 0;
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      y_convolution += frame[i][j] * y_filter[filter_index];
      filter_index++;
    }
  }

  int gradient_magnitude = sqrt((float)((x_convolution * x_convolution) + (y_convolution * y_convolution)));

  to[index] = gradient_magnitude;
}

/* Kernel that runs the edge detection algorithm on each output pixel with
 * sobel filters declared on shared memory.
 *
 * input has width and height that's two bigger than to's because of the way
 * convolution works
 */
__global__ void shared_memory_edge_detection(unsigned char* input, unsigned char* to,
                               int to_size, int to_width, int to_height,
                               int from_width, int from_height) {

  // computes which block this thread lies on
  int blockId = blockIdx.x + (gridDim.x * blockIdx.y);

  // computes the index of the thread
  int index = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.y) + threadIdx.x;

  if (index >= to_size) return;

  // computes values related to param to's 2D frame
  int thread_x = index % to_width;
  int thread_y = index / to_width;

  // frame in original picture needed for convolution
  // size is 3 by 3 because each sobel filter is 3 by 3
  int frame[3][3] = {
    {input[(thread_y) * from_width + (thread_x)], input[(thread_y) * from_width + (thread_x + 1)], input[(thread_y) * from_width + (thread_x + 2)]},
    {input[(thread_y + 1) * from_width + (thread_x)], input[(thread_y + 1) * from_width + (thread_x + 1)], input[(thread_y + 1) * from_width + (thread_x + 2)]},
    {input[(thread_y + 2) * from_width + (thread_x)], input[(thread_y + 2) * from_width + (thread_x + 1)], input[(thread_y + 2) * from_width + (thread_x + 2)]}
  };

  // kernels are now in shared memory
  __shared__ int x_filter[9];
  int x_filter_h[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
  for (int i = 0; i < 9; i++)
    x_filter[i] = x_filter_h[i];
  __shared__ int y_filter[9];
  int y_filter_h[9] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};
  for (int i = 0; i < 9; i++)
    y_filter[i] = y_filter_h[i];

  int x_convolution = 0;
  int filter_index = 0;
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      x_convolution += frame[i][j] * x_filter[filter_index];
      filter_index++;
    }
  }

  int y_convolution = 0;
  filter_index = 0;
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      y_convolution += frame[i][j] * y_filter[filter_index];
      filter_index++;
    }
  }

  int gradient_magnitude = sqrt((float)((x_convolution * x_convolution) + (y_convolution * y_convolution)));

  to[index] = gradient_magnitude;
}

void serial_edge_detection(frame_ptr input, frame_ptr to) {
  for (int y = 0; y < to -> image_height; y++)
    for (int x = 0; x < to -> image_width; x++) {
      // b22 = a13 - a11 + 2a23 - 2a21 + a33 - a31
      // since the input height and width are two more than to's height and width
      // then we need to offset x and y by 1. For example:
      // b00 = a02 - a00 + 2a13 - 2a10 + a22 - a20
            int x_convolution = input->row_pointers[y][x+2] - input->row_pointers[y][x]
	                              + 2 * input->row_pointers[y+1][x+2] - 2 * input->row_pointers[y+1][x]
	      + input->row_pointers[y+2][x+2] - input->row_pointers[y+2][x];

	    // b22 = a11 - 2a12 - a13 + a31 + 2a32 + a33
	    // similar offsets as above
	          int y_convolution = - input->row_pointers[y][x] - 2 * input->row_pointers[y][x+1] - input->row_pointers[y][x+2]
		    + input->row_pointers[y+2][x] + 2 * input->row_pointers[y+2][x+1] + input->row_pointers[y+2][x+2];

		  int gradient_magnitude = sqrt((x_convolution * x_convolution) + (y_convolution * y_convolution));
		  to -> row_pointers[y][x] = gradient_magnitude;
    }
}

void run_cuda_edge_detection(frame_ptr input, frame_ptr to) {
  // size of flattened array (to_components should be 1)
  // input's height and width should both be two more than to's
  int to_size = to->image_height * to->image_width * sizeof(unsigned char);
  int input_size = input->image_height * input->image_width * sizeof(unsigned char);

  unsigned char* input_d; // ptr to input image
  unsigned char* result_d; // ptr to output image
  int* sobel_x_filter_d; // ptr to sobel x filter
  int* sobel_y_filter_d; // ptr to sobel x filter

  // 1. Transfer input to device memory
  // input image
  checkCudaErrors(hipMalloc((void **) &input_d, input_size));
  checkCudaErrors(hipMemcpy(input_d, input->image_buffer, input_size, hipMemcpyHostToDevice));

  // sobel filter
  int x[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
  int y[9] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};
  int (*sobel_x_filter)[9] = &x;
  int (*sobel_y_filter)[9] = &y;
  int filter_size = 9 * sizeof(int);
  checkCudaErrors(hipMalloc((void **) &sobel_x_filter_d, filter_size));
  checkCudaErrors(hipMemcpy(sobel_x_filter_d, sobel_x_filter, filter_size, hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc((void **) &sobel_y_filter_d, filter_size));
  checkCudaErrors(hipMemcpy(sobel_y_filter_d, sobel_y_filter, filter_size, hipMemcpyHostToDevice));

  // Allocate device memory for result
  checkCudaErrors(hipMalloc((void **) &result_d, to_size));

  // 2. Kernel invocation code
  // 32 is the maximum number for this dimension (32, 16, 8, 4, 2)
  int blockDim = 32;
  dim3 DimGrid(to->image_width / blockDim, to->image_height / blockDim, 1);
  // takes ceiling of x and y dimensions
  if (to->image_width % blockDim != 0) DimGrid.x++;
  if (to->image_height % blockDim != 0) DimGrid.y++;

  dim3 DimBlock(blockDim, blockDim, 1);

  // Uses hipEvent_t to get timing information
  hipEvent_t start, stop;
  float time;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  register_memory_edge_detection<<<DimGrid, DimBlock>>>(input_d, result_d, to_size,
                                        to-> image_width, to-> image_height,
                                        input->image_width, input->image_height);

  // global_memory_edge_detection<<<DimGrid, DimBlock>>>(input_d, result_d, to_size,
  //                                        to-> image_width, to-> image_height,
  //                                        input->image_width, input->image_height,
  //                                        sobel_x_filter_d, sobel_y_filter_d);

 // shared_memory_edge_detection<<<DimGrid, DimBlock>>>(input_d, result_d, to_size,
 //                                       to-> image_width, to-> image_height,
 //                                       input->image_width, input->image_height);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  printf("Time run edge detection:  %3.1f ms \n", time);

  // 3. Transfer result from device to host
  checkCudaErrors(hipMemcpy(to->image_buffer, result_d, to_size, hipMemcpyDeviceToHost));

  // Free device memory for input, result
  hipFree(input_d);
  hipFree(result_d);
  hipFree(sobel_x_filter_d);
  hipFree(sobel_y_filter_d);
}


void rgb2grayscale(frame_ptr in, frame_ptr out) {
  unsigned char pixel[3]; // holds rgb value for a given pixel

  for (int row = 0; row < in->image_height; row++) {
    for (int col = 0; col < in->image_width; col++) {
      for (int k=0; k < in->num_components; k++) {
         pixel[k] = in->row_pointers[row][col*in->num_components+k];
       }
       out->row_pointers[row][col] = pixel[0] * 0.3 + pixel[1] * 0.59 + pixel[2] * 0.11;
     }
   }
}


void printPixels(frame_ptr in, char *filename) {

  FILE *fp;
  fp = fopen(filename, "w+");

  for (int i=0; i < in->image_height; i++){
    for (int j=0; j < in->image_width; j++){
      for (int k=0; k < in->num_components; k++){
		     JSAMPLE j1 = in->row_pointers[i][(in->num_components)*j+k];
          fprintf(fp, "%d", j1);
      }
    }
    fprintf(fp, "\n");
  }
  fclose(fp);
}




/*
 * This is just a helper method. It sets the input and output frames
 * and calls edge detection either serially or on cuda
 */
void
runTest( int argc, char** argv)
{

  frame_ptr from = input_frames[0];
  // Allocate frame for kernel to store its results into
  output_frames[0] = allocate_frame(from->image_height, from->image_width, 1);

  frame_ptr testing_frame = allocate_frame(from->image_height, from->image_width, 1);
  //convert image to grayscale
  rgb2grayscale(from, output_frames[0]);

  //printPixels(gray, "gray.txt");

  //pad grayscale image
  frame_ptr img = pad_frame(output_frames[0]);

  //printPixels(img, "padded.txt");

  // call serial algorithm

  // Calculate the time taken by the serial algorithm
  clock_t t;
  t = clock();
  serial_edge_detection(img, testing_frame);
  t = clock() - t;
  double time_taken = ((double)t)/CLOCKS_PER_SEC; // in seconds

  printf("serial_edge_detection() took %f seconds to execute \n", time_taken); 

  // call cuda
  //run_cuda_edge_detection(img, output_frames[0]);

  //checkResults(testing_frame, output_frames[0]);

}

/**
 * Host main routine
 */
int
main(int argc, char **argv)
{

  if(argc < 3){
    usage();
    exit(1);
  }

  // Load input file
  input_frames[0] = read_JPEG_file(argv[1]);

  // Do the actual work including calling CUDA kernel
  runTest(argc, argv);

  // Write output file
  write_JPEG_file(argv[2], output_frames[0], 75);

  return 0;
}
